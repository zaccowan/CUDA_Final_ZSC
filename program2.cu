#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE 10

// CUDA kernel
__global__ void calcR(int* M,int* V, int* R) {
    int i = blockIdx.x;
    int j = threadIdx.x;

    R[i] += M[i * SIZE + j] * V[j];
}

int main(void) {
    int* M;
    int* V;
    int* R;

    hipMalloc(&M, SIZE * SIZE * sizeof(int));
    hipMalloc(&V, SIZE * sizeof(int));
    hipMalloc(&R, SIZE * sizeof(int));

    for( int j = 0; j < SIZE; j++ ) {
        for( int i = 0; i < SIZE; i++ ) {
            M[j * SIZE + i] = i + 1;
        }
    }

    for( int j = 0; j < SIZE; j++ ) {
        V[j] = j + 1;
        R[j] = 0;
    }

    // Print the matrix and vector
    printf("Matrix M:\n");
    for( int j = 0; j < SIZE; j++ ) {
        for( int i = 0; i < SIZE; i++ ) {
            printf("%d ", M[j * SIZE + i]);
        }
        printf("\n");
    }
    printf("Vector V:\n");
    for( int j = 0; j < SIZE; j++ ) {
        printf("%d ", V[j]);
    }
    printf("\n");
    printf("Resultant Vector R before calculation:\n");
    for( int j = 0; j < SIZE; j++ ) {
        printf("%d ", R[j]);
    }
    printf("\n");

    calcR<<<SIZE,SIZE>>>(M, V, R);

    hipDeviceSynchronize();

    // Print the matrix and vector
    printf("Resultant Vector R:\n");
    for( int j = 0; j < SIZE; j++ ) {
        printf("%d ", R[j]);
    }
    printf("\n");

}