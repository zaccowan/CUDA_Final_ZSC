#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE 10

// CUDA kernel
__global__ void calcR(int* M,int* V, int* R) {
    int i = blockIdx.x;
    int j = threadIdx.x;

    R[i] += M[i * SIZE + j] * V[j];
}

int main(void) {
    int* M;
    int* V;
    int* R;

    // Allocate memory + Syncrohnize between CPU and GPU
    hipMalloc(&M, SIZE * SIZE * sizeof(int));
    hipMalloc(&V, SIZE * sizeof(int));
    hipMalloc(&R, SIZE * sizeof(int));


    // Fill the matrix where a given element is the column index + 1
    for( int j = 0; j < SIZE; j++ ) {
        for( int i = 0; i < SIZE; i++ ) {
            M[j * SIZE + i] = i + 1;
        }
    }
    
    // Fill the vector, V, where a given element is the index + 1
    // Initialize the resultant vector, R, to 0's for all elements
    for( int j = 0; j < SIZE; j++ ) {
        V[j] = j + 1;
        R[j] = 0;
    }

    // Print the matrix and vector
    printf("Matrix M:\n");
    for( int j = 0; j < SIZE; j++ ) {
        for( int i = 0; i < SIZE; i++ ) {
            printf("%d ", M[j * SIZE + i]);
        }
        printf("\n");
    }
    printf("Vector V:\n");
    for( int j = 0; j < SIZE; j++ ) {
        printf("%d ", V[j]);
    }
    printf("\n");
    printf("Resultant Vector R before calculation:\n");
    for( int j = 0; j < SIZE; j++ ) {
        printf("%d ", R[j]);
    }
    printf("\n");

    // Call the cuda specific function to calculate the resultant vector
    calcR<<<SIZE,SIZE>>>(M, V, R);

    hipDeviceSynchronize();

    // Print the matrix and vector
    printf("Resultant Vector R:\n");
    for( int j = 0; j < SIZE; j++ ) {
        printf("%d ", R[j]);
    }
    printf("\n");

}