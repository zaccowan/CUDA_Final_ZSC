#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE 10

// CUDA kernel
__global__ void calcMatrixElement(int* M,int* V) {

    int i = blockIdx.x;
    int j = threadIdx.x;

    M[i * SIZE + j] = M[i * SIZE + j] * V[j];
}

int main(void) {
    int* M;
    int* V;

    // Allocate memory + Syncrohnize between CPU and GPU
    hipMalloc(&M, SIZE * SIZE * sizeof(int));
    hipMalloc(&V, SIZE * sizeof(int));

    // Fill the matrix where a given element is the column index + 1
    for( int j = 0; j < SIZE; j++ ) {
        for( int i = 0; i < SIZE; i++ ) {
            M[j * SIZE + i] = i + 1;
        }
    }

    // Fill the vector where a given element is the index + 1
    for( int j = 0; j < SIZE; j++ ) {
        V[j] = j + 1;
    }

    // Print the matrix and vector
    printf("Matrix M:\n");
    for( int j = 0; j < SIZE; j++ ) {
        for( int i = 0; i < SIZE; i++ ) {
            printf("%d ", M[j * SIZE + i]);
        }
        printf("\n");
    }
    printf("Vector V:\n");
    for( int j = 0; j < SIZE; j++ ) {
        printf("%d ", V[j]);
    }
    printf("\n");


    // Call the cuda specific function to calculate the resultant matrix
    // Each thread will calculate one element of the resultant matrix
    calcMatrixElement<<<SIZE,SIZE>>>(M, V);

    hipDeviceSynchronize();

    // Print the matrix and vector
    printf("Resultant Matrix M:\n");
    for( int j = 0; j < SIZE; j++ ) {
        for( int i = 0; i < SIZE; i++ ) {
            printf("%d ", M[j * SIZE + i]);
        }
        printf("\n");
    }

    hipFree(M);
    hipFree(V);

}