#include <stdio.h>
#include <hip/hip_runtime.h>

#define ARRAY_SIZE 10

// CUDA kernel
__global__ void calcMatrixElement() {
}

int main(void) {
    int* M;
    int* V;

    M = (int*)malloc(ARRAY_SIZE * ARRAY_SIZE * sizeof(int));
    V = (int*)malloc(ARRAY_SIZE * sizeof(int));

    hipMalloc(&M, ARRAY_SIZE * ARRAY_SIZE * sizeof(char));
    hipMalloc(&V, ARRAY_SIZE * sizeof(char));

    for( int j = 0; j < ARRAY_SIZE; j++ ) {
        for( int i = 0; i < ARRAY_SIZE; i++ ) {
            M[j * ARRAY_SIZE + i] = i + 1;
        }
    }

    for( int j = 0; j < ARRAY_SIZE; j++ ) {
        V[j] = j + 1;
    }

    // Print the matrix and vector
    printf("Matrix M:\n");
    for( int j = 0; j < ARRAY_SIZE; j++ ) {
        for( int i = 0; i < ARRAY_SIZE; i++ ) {
            printf("%d ", M[j * ARRAY_SIZE + i]);
        }
        printf("\n");
    }
    printf("Vector V:\n");
    for( int j = 0; j < ARRAY_SIZE; j++ ) {
        printf("%d ", V[j]);
    }

}